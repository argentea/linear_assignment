
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iostream>
#include <string>
#include <fstream>
#include <hip/hip_runtime_api.h>

#include <stdio.h>
#include <stdlib.h>
#include <vector>

#include <chrono>

// --
// Define constants

#define DEBUG   1
#define DEBUG_SHOW  5

#ifndef __RUN_VARS
#define __RUN_VARS
#define AUCTION_MAX_EPS 10.0 // Larger values mean solution is more approximate
#define AUCTION_MIN_EPS 1.0
#define AUCTION_FACTOR  0.1
#define NUM_RUNS        1
#define BATCH_SIZE     1024
#define MAX_ITERATIONS  500
#define NUM_NODES 128
#define BIG_NEGATIVE -9999999
#endif


typedef std::chrono::high_resolution_clock::rep hr_clock_rep;

inline hr_clock_rep get_globaltime(void) 
{
	using namespace std::chrono;
	return high_resolution_clock::now().time_since_epoch().count();
}

// Returns the period in miliseconds
inline double get_timer_period(void) 
{
	using namespace std::chrono;
	return 1000.0 * high_resolution_clock::period::num / high_resolution_clock::period::den;
}


template <typename T>
__global__ void __launch_bounds__(1024, 16)
linear_assignment_auction_kernel(const int num_nodes,
                                const T* __restrict__ cost_ptr,
                                int* solution_ptr, 
                                float*  bids_ptr,
                                char* stop_flag_ptr,
                                const float auction_max_eps,
                                const float auction_min_eps,
                                const float auction_factor,
                                const int max_iterations)
{
    const int batch_id = blockIdx.x;
    const int node_id = threadIdx.x;

    int local_front_edge_count = 0;
    int local_edge_count = 0;

    __shared__ float auction_eps;
    __shared__ int num_iteration;
    __shared__ int num_assigned;
    __shared__ int num_edge;

    extern __shared__ unsigned char s_data[];
    T* prices = (T*)s_data;
    int* sbids = (int*)(prices + num_nodes);
    int* person2item = sbids + num_nodes;
    int* item2person = person2item + num_nodes;

    unsigned char* edge_count = (unsigned char*)(item2person + num_nodes);
    unsigned char* item_id = (unsigned char*)(edge_count + num_nodes);
    unsigned short* benefit;

    if(node_id == 0){
        auction_eps = auction_max_eps;
        num_iteration = 0;
    }

    const T* __restrict__ data = cost_ptr + batch_id * num_nodes * num_nodes;
    int* solution_global = solution_ptr + batch_id * num_nodes; 
    float* bids = bids_ptr + batch_id * num_nodes * num_nodes;
    char* stop_flag = stop_flag_ptr + batch_id;
    
    prices[node_id] = 0;

    __syncthreads();
    //count the items connected to bidder
    edge_count[node_id] = 0;

    for(int i = 0; i < num_nodes; i++){
        if(data[node_id * num_nodes + i] >= 0)
            edge_count[node_id]++;
    } 
    __syncthreads();

    if(DEBUG && 0){
        if(batch_id==2){
            printf("%d %d\n",node_id,edge_count[node_id]);
        }
        __syncthreads();
    }

    //that's can be optmized

    for(int i = 0; i < node_id; i++){
        local_front_edge_count += edge_count[i];
    }
    
    //Is that faster than read from share_memory?
    local_edge_count = edge_count[node_id];

    if(node_id == num_nodes -1){
        num_edge = edge_count[node_id - 1] + local_front_edge_count;
    }
    __syncthreads();

    benefit = (unsigned short*)(item_id + num_edge);

    int tem = 0;
    for(int i = 0; i < num_nodes; i++){
        if(data[node_id * num_nodes + i] >= 0){
            item_id[local_front_edge_count + tem] = i;
            benefit[local_front_edge_count + tem] = data[node_id*num_nodes + i];
            tem++;
        }
    }
    __syncthreads();


    if(DEBUG && 0){
        if(batch_id == 2){
            printf("%d %d\n",num_edge,node_id);
        }
    }

    if(DEBUG && 0){
        if(batch_id==2){
            printf("%d %d\n",node_id,local_front_edge_count);
        }
        __syncthreads();
    }


    /*
    int tem_count = 0;
    for(int i = 0; i < num_nodes; i++){
        if(data[node_id * num_nodes + i] >= 0){
            local_edges[tem_count].item_id = i;
            local_edges[tem_count].value = data[node_id * num_nodes + i];
            tem_count++;
        }
        else{
            continue;
        }
        //that's may be faster
        if(tem_count >= local_edge_count){
            break;
        }
    }
    __syncthreads();
    */
    while(auction_eps >= auction_min_eps && num_iteration < max_iterations)
    {
        //clear num_assigned
        if(node_id == 0){
            num_assigned = 0;
        }

        //pre-init
        person2item[node_id] = -1;
        item2person[node_id] = -1;
        
        __syncthreads();
        //start iterative solving
        while(num_assigned < num_nodes && num_iteration < max_iterations)
        {
            //phase 1: init bid and bids
            
            for(int i = node_id; i < num_nodes*num_nodes; i += blockDim.x){
                bids[i] = 0;
            }
            sbids[node_id] = 0;
            
            __syncthreads();

            //phase 2: bidding
            if(person2item[node_id] == -1){
                float top1_val = BIG_NEGATIVE; 
                float top2_val = BIG_NEGATIVE; 
                int top1_col; 
                unsigned char tem_id;
                float tmp_val;
                #pragma unroll 32
                for (int i = 0; i < local_edge_count; i++)
                {
                    tem_id = item_id[local_front_edge_count + i];
                    tmp_val = benefit[local_front_edge_count + i] - prices[tem_id]; 
                    if (tmp_val >= top1_val)
                    {
                        top2_val = top1_val;
                        top1_col = tem_id;
                        top1_val = tmp_val;
                    }
                    else if (tmp_val > top2_val)
                    {
                        top2_val = tmp_val;
                    }
                }
                if (top2_val == BIG_NEGATIVE)
                {
                    top2_val = top1_val;
                }
                float bid = top1_val - top2_val + auction_eps;
                
                atomicMax(sbids+top1_col, 1);
                bids[num_nodes * top1_col + node_id] = bid;
                
            }

            __syncthreads();

            //phase 3 : assignment
            if(sbids[node_id] != 0) {
                float high_bid  = 0;
                int high_bidder = -1;
    
                float tmp_bid = -1;
                #pragma unroll 64
                for(int i = 0; i < num_nodes; i++){
                    tmp_bid = bids[node_id * num_nodes + i];
                    if(tmp_bid > high_bid){
                        high_bid    = tmp_bid;
                        high_bidder = i;
                    }
                }
    
                int current_person = item2person[node_id];
                if(current_person >= 0){
                    person2item[current_person] = -1;
                } else {
                    atomicAdd(&num_assigned, 1);
                }
    
                prices[node_id]                += high_bid;
                person2item[high_bidder]          = node_id;
                item2person[node_id]              = high_bidder;
            }
            
            //update iteration
            if(node_id == 0){
                num_iteration++;
            }
            __syncthreads();
        }
        //scale auction_eps
        if(node_id == 0){
            auction_eps *= auction_factor;
        }
        __syncthreads();
    }
    __syncthreads();
    //report whether finish solving
    if(node_id == 0){
        *stop_flag = (num_assigned == num_nodes);
    }
    //write result out
    
    solution_global[node_id] = person2item[node_id];
    
}

template <typename T>
void linear_assignment_auction(
                const T* cost_matrics,
                int* solutions,
                const int num_graphs,
                const int num_nodes,
                char* scratch,
                char *stop_flags,
                float auction_max_eps,
                float auction_min_eps,
                float auction_factor,
                int max_iterations)
{
    //get pointers from scratch (size: num_nodes*num_nodes*sizeof(T))
    float* bids           = (float* )scratch;

    //launch solver
    hipProfilerStart();
    linear_assignment_auction_kernel<T><<<num_graphs, num_nodes, ((num_nodes)*num_nodes)*sizeof(T)/3>>>
                                    (
                                        num_nodes,
                                        cost_matrics,
                                        solutions,
                                        bids,
                                        stop_flags,
                                        auction_max_eps,
                                        auction_min_eps,
                                        auction_factor,
                                        max_iterations
                                    );
    hipProfilerStop();
    hipDeviceSynchronize();

}

hr_clock_rep timer_start, timer_stop;

template <typename T>
void run_auction(
    int    num_graphs,
    int    num_nodes,
    T* h_data,      // data
    int*   h_person2item[], // results
    float auction_max_eps,
    float auction_min_eps,
    float auction_factor,
    int num_runs,
    int verbose
)
{
    T *data;
    char* scratch;
    int* solutions;
    char* stop_flags;

    hipMalloc((void **)&data,          BATCH_SIZE * num_nodes*num_nodes   * sizeof(T));
    hipMalloc((void**) &scratch, num_graphs*(num_nodes*num_nodes)*sizeof(float));
    hipMalloc((void**)& solutions, num_graphs*num_nodes*sizeof(int));
    hipMalloc((void**)& stop_flags, sizeof(char) * num_graphs);

    hipMemcpy(data, h_data, num_graphs* num_nodes*num_nodes* sizeof(T), hipMemcpyHostToDevice);
    
    timer_start = get_globaltime();

    linear_assignment_auction<T>(data,
                                solutions,
                                num_graphs,
                                num_nodes,
                                scratch,
                                stop_flags,
                                auction_max_eps,
                                auction_min_eps,
                                auction_factor,
                                MAX_ITERATIONS);

    hipDeviceSynchronize();
    timer_stop = get_globaltime();
    

    for (int i = 0; i < BATCH_SIZE; ++i)
    {
        hipMemcpy(h_person2item[i], solutions + i * num_nodes, sizeof(int) * num_nodes, hipMemcpyDeviceToHost);
    }

    hipFree(data);
    hipFree(scratch);
    hipFree(solutions);
    hipFree(stop_flags);
    return;
}


template <typename T>
int load_data(T *raw_data) {
    std::ifstream input_file("graph4", std::ios_base::in);

    int i = 0;
    T val;
    while(input_file >> val) {
        raw_data[i] = val;
        i++;
        
    }
    return (int)sqrt(i);
}

int main(int argc, char **argv)
{

    std::cerr << "loading ./graph4" << std::endl;
    int num_nodes = NUM_NODES;
    int num_graphs = BATCH_SIZE;
    int *h_data = new int[num_graphs*num_nodes*num_nodes];
    int* h_person2item[BATCH_SIZE];

    
    for (int i = 0; i < BATCH_SIZE; ++i)
    {
        num_nodes = load_data<int>(h_data + i*num_nodes*num_nodes);
        h_person2item[i] = (int *)malloc(sizeof(int) * num_nodes);
    }

    int verbose = 1;
    
    run_auction<int>(
        num_graphs,
        num_nodes,
        h_data,
        h_person2item,
        AUCTION_MAX_EPS,
        AUCTION_MIN_EPS,
        AUCTION_FACTOR,
        NUM_RUNS,
        verbose
    );
    
    

    // // Print results
    for (int i = 0; i < 1; ++i)
    {
        std::cerr << "solution " << i << "\n";
        for (int j = 0; j < num_nodes; j++) {
            std::cerr << j << ":" << h_person2item[i][j] << ", "; 
        }
        std::cerr << std::endl; 

        float score = 0;
        for (int j = 0; j < num_nodes; j++) {
            score += h_data[i*num_nodes*num_nodes+j * num_nodes + h_person2item[i][j]];
        }

        std::cerr << "score=" << (int)score << std::endl;

    }
    delete[] h_data;
    std::cerr << "[D] run_auction takes "<< (timer_stop-timer_start)*get_timer_period() <<  "ms\n";
    //printf("[D] run_auction takes %g ms\n", (timer_stop-timer_start)*get_timer_period()); 
}
